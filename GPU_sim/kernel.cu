#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#include "RSParser.h"
#include "optionparser.h"

#include <stdio.h>
#include <math.h>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <map>

__constant__ unsigned int DEV_CONST_REACTIONS[8000];
__constant__ unsigned int DEV_CONST_OFFSET[8000];

template <class T>
inline T minimo( T v1, T v2 ) {
	return
		v1 > v2? v2 : v1;		
}

template <typename T, typename T2>
inline T minimo( T v1, T2 v2 ) {
	return
		v1 > v2? v2 : v1;		
}

struct Arg: public option::Arg
{
  static void printError(const char* msg1, const option::Option& opt, const char* msg2)
  {
    fprintf(stderr, "%s", msg1);
    fwrite(opt.name, opt.namelen, 1, stderr);
    fprintf(stderr, "%s", msg2);
  }

  static option::ArgStatus Required1(const option::Option& option, bool msg)
  {
    if (option.arg != 0)
      return option::ARG_OK;

    if (msg) printError("Option '", option, "' requires an argument\n");
    return option::ARG_ILLEGAL;
  }
  
  static option::ArgStatus Required2(const option::Option& option, bool msg)
  {
    if (option.arg != 0)
      return option::ARG_OK;

    if (msg) printError("Option '", option, "' requires an argument\n");
    return option::ARG_ILLEGAL;
  }

  static option::ArgStatus Required3(const option::Option& option, bool msg)
  {
    if (option.arg != 0)
      return option::ARG_OK;

    if (msg) printError("Option '", option, "' requires an argument\n");
    return option::ARG_ILLEGAL;
  }

  static option::ArgStatus Numeric1(const option::Option& option, bool msg)
  {
    char* endptr = 0;
    if (option.arg != 0 && strtol(option.arg, &endptr, 10)){};
    if (endptr != option.arg && *endptr == 0)
      return option::ARG_OK;

    if (msg) printError("Option '", option, "' requires a numeric argument\n");
    return option::ARG_ILLEGAL;
  }

    static option::ArgStatus Numeric2(const option::Option& option, bool msg)
  {
    char* endptr = 0;
    if (option.arg != 0 && strtol(option.arg, &endptr, 10)){};
    if (endptr != option.arg && *endptr == 0)
      return option::ARG_OK;

    if (msg) printError("Option '", option, "' requires a numeric argument\n");
    return option::ARG_ILLEGAL;
  }

};

void dump_grezzo_reazioni(ReactionSystemsParser* rsp) {

	for (unsigned int r=0; r<rsp->get_number_of_reactions(); r++ ) {
		printf(" * Reazione %u: ", r);
		unsigned int pos = rsp->vettore_offset[r];
		while( rsp->vettore_regole[pos]!=FINE_REAZIONE ) {
			printf("%u, ", rsp->vettore_regole[pos]);
			pos++;
		}
		printf("\n");
	}
	printf("\n");

}

void dump_grezzo_stato_iniziale(ReactionSystemsParser* rsp) {
	printf(" * Dumping initial state of the RS:\n");
	for (unsigned int s=0; s<rsp->get_number_of_species(); s++) {
			printf("%u\t", rsp->vettore_stati[s]);
		}
	printf("\n");
}

void start_profiling(hipEvent_t* start, hipEvent_t* stop) {

	/// TIMER 1	
	hipEventCreate(start);  
	hipEventCreate(stop);
	hipEventRecord(*start, 0);

}

float stop_profiling(hipEvent_t* start, hipEvent_t* stop) {

	hipEventRecord( *stop, 0 );
	hipEventSynchronize( *stop );
	float tempo;
	hipEventElapsedTime( &tempo, *start, *stop );
	tempo /= 1000;
	// printf("Tempo di esecuzione: %f s\n", tempo);
	return tempo;

}

template <bool use_context>
__global__ void Context( char* Stato, unsigned int num_stato, unsigned int num_sostanze, char* dev_context, unsigned int step ) {

	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if ( blockIdx.x == gridDim.x-1 ) {
		if (tid>num_sostanze-1) return;
	}

	char stato_corrente = num_stato;
	char stato_prossimo = num_stato ^ 1;

	// reset the next state for results
	Stato[ tid + stato_prossimo*num_sostanze ] = 0;

	// overwrite the CURRENT state with context
	if (use_context) {
		char val = dev_context[tid+step*num_sostanze];
		if (val==1) Stato[ tid + stato_corrente*num_sostanze ] = val;
	} 

//	printf("TID %d current state %d.\n", tid,   Stato[ tid + stato_corrente*num_sostanze ]);
//	printf("TID %d next state %d.\n", tid,		Stato[ tid + stato_prossimo*num_sostanze ]);

}

template<bool use_const>
__global__ void Simulate_Lightweight( 
	const unsigned int* Reazioni, char* Stato, const unsigned int* offset, 
	const unsigned int num_stato, const unsigned int reazioni, const unsigned int num_sostanze) {

	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if ( blockIdx.x == gridDim.x-1 ) {
		if (tid>reazioni-1) return;
	}

	unsigned int pos;
	if (use_const) 
		pos = DEV_CONST_OFFSET[tid];
	else
		pos = offset[tid];

	char stato_corrente = num_stato ^ 1;
	char stato_prossimo = num_stato ;

	bool reagente = Stato[ stato_corrente*num_sostanze + DEV_CONST_REACTIONS[pos] ];
	bool inibitore = Stato[ stato_corrente*num_sostanze + DEV_CONST_REACTIONS[pos+2] ];			

	if ( reagente && (!inibitore) ) {
		Stato[ stato_prossimo*num_sostanze + DEV_CONST_REACTIONS[pos+4] ] = 1;
	} else {
		Stato[ stato_prossimo*num_sostanze + DEV_CONST_REACTIONS[pos+4] ] = 0;
	}
	
}

template<bool use_const>
__global__ void Simulate( 
	const unsigned int* Reazioni, char* Stato, const unsigned int* offset, 
	const unsigned int num_stato, const unsigned int reazioni, const unsigned int num_sostanze) {

	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if ( blockIdx.x == gridDim.x-1 ) {
		if (tid>reazioni-1) return;
	}

	unsigned int pos;
	if (use_const) 
		pos = DEV_CONST_OFFSET[tid];
	else
		pos = offset[tid];

	char stato_corrente = num_stato ^ 1;
	char stato_prossimo = num_stato ;

	bool risultato = true;


	// processiamo i reagenti
	if (use_const) {
		while(DEV_CONST_REACTIONS[pos]!=FINE_REAGENTI) {		
			risultato &= Stato[ stato_corrente*num_sostanze + DEV_CONST_REACTIONS[pos] ];			
			pos++;
		}
	} else {
		while(Reazioni[pos]!=FINE_REAGENTI) {		
			risultato &= Stato[ stato_corrente*num_sostanze + Reazioni[pos] ];
			pos++;
		}
	}

//	printf("[Reactants] TID: %d result %d.\n", tid, risultato);
	
	pos++;

	// processiamo gli inibitori
	if (use_const) {
		while(DEV_CONST_REACTIONS[pos]!=FINE_INIBITORI) {
			risultato &= !Stato[ stato_corrente*num_sostanze + DEV_CONST_REACTIONS[pos] ];
			pos++;
		}
	} else {
		while(Reazioni[pos]!=FINE_INIBITORI) {
			risultato &= !Stato[ stato_corrente*num_sostanze + Reazioni[pos] ];
			pos++;
		}
	}

//	printf("[Inhibitors] TID: %d result %d.\n", tid, risultato);

	pos++;

	// calcoliamo i risultati
	if (risultato) {
		if (use_const) {
			while(DEV_CONST_REACTIONS[pos]!=FINE_REAZIONE) {
				Stato[ stato_prossimo*num_sostanze +  DEV_CONST_REACTIONS[pos] ] = 1;
				pos++;				
			}
		} else {
			while(Reazioni[pos]!=FINE_REAZIONE) {
				Stato[ stato_prossimo*num_sostanze +  Reazioni[pos] ] = 1;				
				// printf("[Update (res=1)] TID: %d result %d.\n", tid, Stato[ stato_prossimo*num_sostanze +  Reazioni[pos] ]);
				pos++;
			}
		}
	} else {
		// printf("[Update (res=0)] TID: %d result %d.\n", tid, Stato[ stato_prossimo*num_sostanze +  Reazioni[pos] ]);
	}
}


__global__ void SaveTrace(char* state, char* trace, const long unsigned int step, const long unsigned int species, const unsigned int numstato) {

	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid>species-1) return;	

	trace[ species*step + tid ] = state[ numstato*species + tid ] ; 
	// printf("[Trace] TID %d state %d.\n", tid, trace[species*step + tid ]);

}


unsigned int numBlocchi( unsigned int NUM_THREADS, unsigned int TPB ) {

	return   NUM_THREADS / TPB + 1;
	
}


void calculateGroupsAndStates(const size_t est, const size_t freem, const unsigned int species, unsigned long int* groups, unsigned long int* statesPerGroup) {
	unsigned long int half = floorl(freem/2);
	*statesPerGroup = floorl( half/((sizeof(char))*species) );
	*groups = floorl( est / (*statesPerGroup) ) + 1;
}

enum  optionIndex { HELP, REQUIRED1, REQUIRED2, REQUIRED3, REQUIRED4, NUMERIC1, NUMERIC2, CONSMEMORY, VERBOSE, LIGHTWEIGHT };
const option::Descriptor usage[] = {
	{ HELP,    0,"", "help",    Arg::None,    "  \t--help  \tPrint usage and exit." },
	{ REQUIRED1, 0,"r","rules",Arg::Required1,"  -r <arg>, \t--rules=<arg>  \tInput file specifying the rules." },
	{ REQUIRED2, 0,"i","initial",Arg::Required2,"  -i <arg>, \t--initial=<arg>  \tInput file specifying the initial state of the system." },
	{ REQUIRED3, 0,"o","output",Arg::Required2,"  -o <arg>, \t--output=<arg>  \tInput file specifying the output file of the trace." },
	{ REQUIRED4, 0,"c","context",Arg::Required2,"  -c <arg>, \t--context=<arg>  \tInput file specifying the context of the reaction system." },
	{ NUMERIC1, 0,"s","steps", Arg::Numeric1, "  -s <num>, \t--steps=<num>  \tRequires a number as argument." },
	{ NUMERIC2, 0,"b","blocks", Arg::Numeric2, "  -b <num>, \t--blocks=<num>  \tRequires a number as argument." },
	{ CONSMEMORY, 0,"n","no_constant", Arg::None, "  -n, \t--no_constant  \tDisables the constant memory." },
	{ VERBOSE, 0,"v","verbose", Arg::None, "  -v, \t--verbose\tEnables verbose mode." },
	{ LIGHTWEIGHT, 0,"l","lightweight", Arg::None, "  -l, \t--lightweight\tEnables lightweight kernel for normal systems." },
	{ 0, 0, 0, 0, 0, 0 } };


int main(int argc, char**argv )
{
	argc-=(argc>0); argv+=(argc>0); // skip program name argv[0] if present
	option::Stats stats(usage, argc, argv);

	#ifdef __GNUC__
		// GCC supports C99 VLAs for C++ with proper constructor calls.
		option::Option options[stats.options_max], buffer[stats.buffer_max];
	#else
		// use calloc() to allocate 0-initialized memory. It's not the same
		// as properly constructed elements, but good enough. Obviously in an
		// ordinary C++ program you'd use new[], but this file demonstrates that
		// TLMC++OP can be used without any dependency on the C++ standard library.
		option::Option* options = (option::Option*)calloc(stats.options_max, sizeof(option::Option));
		option::Option* buffer  = (option::Option*)calloc(stats.buffer_max,  sizeof(option::Option));
	#endif

	option::Parser parse(usage, argc, argv, options, buffer);

	if (parse.error())    return 1;
	if (options[HELP] || argc == 0)
	{
		int columns = getenv("COLUMNS")? atoi(getenv("COLUMNS")) : 80;
		option::printUsage(fwrite, stdout, usage, columns);
		return 0;
	}

	std::string input_path("");
	std::string state_path("");
	std::string output_path("");
	std::string context_path("");
	unsigned long int MAX_PASSI = 0;
	unsigned int DIM_BLOCCO = 32;
	bool use_constant = false;
	bool use_context  = false;
	bool verbose = false;
	bool force_disable_constant_memory  = false;
	bool lightweight = false;
	bool output_to_console = true;

	for (int i = 0; i < parse.optionsCount(); ++i)
	{
		option::Option& opt = buffer[i];
		// fprintf(stdout, "Argument #%d is ", i);
		switch (opt.index())
		{
		  case HELP:
			// not possible, because handled further above and exits the program
		  case REQUIRED1:
			input_path = opt.arg;
			break;
		  case REQUIRED2:
			state_path = opt.arg;
			break;
		  case REQUIRED3:			
			output_path = opt.arg;
			break;
		  case REQUIRED4:			
			context_path = opt.arg;
			break;
		  case NUMERIC1:
			MAX_PASSI = atoi(opt.arg);
			break;		  
		  case NUMERIC2:			
			DIM_BLOCCO = atoi(opt.arg);
			break;		  
		  case CONSMEMORY:
			force_disable_constant_memory = true;
			break;
		  case VERBOSE:			
			verbose = true;
			break;
		  case LIGHTWEIGHT:			
			lightweight = true;
			break;
		 /* case UNKNOWN:
			// not possible because Arg::Unknown returns ARG_ILLEGAL
			// which aborts the parse with an error
			break; */
		}
  }

	if (verbose) {
		fprintf(stdout, " * Verbose mode enabled.\n");
		fprintf(stdout, " * RS rules loaded from file: '%s'\n", input_path);
		fprintf(stdout, " * Output file: '%s'\n", output_path);
		fprintf(stdout, " * Context file: '%s'\n", context_path);
		// fprintf(stdout, " * Initial state loaded from file:  '%s'\n", state_path);
		if (force_disable_constant_memory) fprintf(stdout, " * Constant memory disabled.\n");
		if (lightweight) fprintf(stdout, " * Lightweight kernel enabled.\n");
		// fprintf(stdout, " * --blocks with argument '%s'\n", DIM_BLOCCO);
		// fprintf(stdout, " * Simulation steps: %s\n", MAX_PASSI);		
	}




	ReactionSystemsParser rsp;
	// rsp.OpenFile(argv[1], argv[2]);	
	rsp.OpenFile(input_path, state_path, context_path);

	if (rsp.vector_context.size()>0) use_context = true;
	if (use_context && verbose) printf(" * Using context.\n");

	/* arguments: 
		- rules file
		- initial state file
		- running steps
		- block dimension
	*/

	// query device (0) for properties
	hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

	// force number of steps?
	if (MAX_PASSI==0) {
		MAX_PASSI = rsp.get_iterations();				
	} else {
		MAX_PASSI = minimo(MAX_PASSI, rsp.get_iterations());		
	}
	if (verbose) fprintf(stdout, " * Simulation will perform %d iterations.\n", MAX_PASSI);

	unsigned long int estimatedGM;
	// unsigned long int bytes_per_stream;
	estimatedGM = MAX_PASSI * (unsigned long int)(rsp.get_number_of_species()) * (unsigned long int)(sizeof(char)) ;

	size_t freem, total;
	hipMemGetInfo(&freem, &total);  

	if (verbose) {
		printf(" * Total global memory: %lu\n", total);
		printf(" * Available global memory: %lu\n", freem);
		printf(" * Estimation of the global memory needed to store the dynamics of the RS: %lu\n", estimatedGM);
		fprintf(stdout, " * Number of detected chemical species: %d.\n", rsp.get_number_of_species() );
		fprintf(stdout, " * Number of detected reactions: %d.\n", rsp.get_number_of_reactions() );
	}

	unsigned long int groups = 0;
	unsigned long int statesPerGroup = 0;

	if ( freem < estimatedGM )  {
		//printf("WARNING: output file is larger than GPU's global memory: using multiple streams.\n");		
		/*
			Dividiamo l'output in due gruppi: stream 0 e stream 1 (lettura durante simulazione).
			I gruppi sono spaccati in pi� sottoinsiemi sequenziali.
			Dobbiamo determinare quante iterazioni copre ogni gruppo.
		*/		
		calculateGroupsAndStates(estimatedGM, freem, rsp.get_number_of_species(), &groups, &statesPerGroup);		
	} else {
		groups = 1;
		statesPerGroup = MAX_PASSI;
	}

	if (verbose) fprintf(stdout, " * Using %lu groups with %lu states\n", groups, statesPerGroup );


	// TODO
	groups = 1;
	statesPerGroup = MAX_PASSI;
	if (verbose) fprintf(stdout, "WARNING: multiple groups and streams disabled.\n");


	char* dev_results[2];	
	char* host_results[2];	

	if (groups==1) {
		hipMalloc(&dev_results[0], sizeof(char)*rsp.get_number_of_species() );
		host_results[0] = (char*) malloc ( sizeof(char) * rsp.get_number_of_species() );		
	} else {
		hipMalloc(&dev_results[0], sizeof(char)*statesPerGroup );
		hipMalloc(&dev_results[1], sizeof(char)*statesPerGroup );
		host_results[0] = (char*) malloc ( sizeof(char)*statesPerGroup );
		host_results[1] = (char*) malloc ( sizeof(char)*statesPerGroup );
	}
				
	if (verbose) printf(" * Requested %u blocks\n", numBlocchi(rsp.get_number_of_species(), DIM_BLOCCO) );

	char* dev_stato;
	unsigned int* dev_regole;
	unsigned int* dev_offset;
	char* read_back = (char*) malloc ( sizeof(char)*rsp.get_number_of_species()*2 );

	hipMalloc( &dev_stato, sizeof(char) * rsp.get_number_of_species() * 2 );	
	hipMalloc( &dev_regole, sizeof(unsigned int) * rsp.vettore_regole.size() );
	hipMalloc( &dev_offset, sizeof(unsigned int) * rsp.vettore_offset.size() );

	if (verbose) fprintf(stdout, " * Rules vector requires %d bytes.\n", sizeof(unsigned int) * rsp.vettore_regole.size() );
	if (verbose) fprintf(stdout, " * Offsets vector requires %d bytes.\n", sizeof(unsigned int) * rsp.vettore_offset.size() );
	if ( sizeof(unsigned int) * rsp.vettore_regole.size() < 8000) {
		if (verbose) fprintf(stdout, " * It could possible to exploit the constant memory to store the model.\n");
		unsigned int model_size = sizeof(unsigned int) * rsp.vettore_regole.size() ;
		hipMemcpyToSymbol(HIP_SYMBOL(DEV_CONST_REACTIONS), &(rsp.vettore_regole[0]), sizeof(unsigned int) * rsp.vettore_regole.size()); 
		hipMemcpyToSymbol(HIP_SYMBOL(DEV_CONST_OFFSET), &(rsp.vettore_offset[0]), sizeof(unsigned int) * rsp.vettore_offset.size()); 
		use_constant = true;
		if (force_disable_constant_memory) {
			use_constant=false; // override
			if (verbose) fprintf(stdout, "WARNING: constant memory disabled.\n");
		}
	}
	
	// memory context
	char* dev_context;
	// unsigned int* dev_context_offset;
	hipMalloc( &dev_context, sizeof(char)*rsp.vector_context.size() );
	// hipMalloc( &dev_context_offset, sizeof(unsigned int)*rsp.vector_context_offset.size() );
	hipMemcpy( dev_context, &(rsp.vector_context[0]), sizeof(char)*rsp.vector_context.size() , hipMemcpyHostToDevice );
	// hipMemcpy( dev_context_offset, &(rsp.vector_context_offset[0]), sizeof(unsigned int)*rsp.vector_context_offset.size() , hipMemcpyHostToDevice );
	
	hipMemcpy( dev_regole, &(rsp.vettore_regole[0]), sizeof( unsigned int )*rsp.vettore_regole.size(), hipMemcpyHostToDevice );
	hipMemcpy( dev_offset, &(rsp.vettore_offset[0]), sizeof( unsigned int )*rsp.vettore_offset.size(), hipMemcpyHostToDevice );	
	hipMemcpy( dev_stato,  &(rsp.vettore_stati[0]),  sizeof( char ) * rsp.get_number_of_species()*2 , hipMemcpyHostToDevice );

	// memoria per la traccia
	char* host_trace = (char*) malloc ( sizeof(char) * rsp.get_number_of_species() * MAX_PASSI );
	char* dev_trace;
	memset(host_trace, 0, sizeof(char) * rsp.get_number_of_species() * MAX_PASSI);
	hipMalloc( &dev_trace, sizeof(char) * rsp.get_number_of_species() * MAX_PASSI );
	hipMemcpy( dev_trace, host_trace, sizeof(char) * rsp.get_number_of_species() * MAX_PASSI, hipMemcpyHostToDevice );
	
	// Profiling 
	hipEvent_t start,  stop;
	start_profiling(&start, &stop);

	//printf(" * Initial state loaded from input files:\n");
	if (verbose)  {
		for (unsigned int i=0; i<rsp.get_number_of_species()*2; i++) {
			printf("Species %d: %d\t", i, rsp.vettore_stati.at(i));
		}
		printf("\n\n");
	}	

	unsigned int num_stato = 0;

	/*
	if (use_context) 
		Context<true><<< numBlocchi(rsp.get_number_of_species(), DIM_BLOCCO), DIM_BLOCCO>>>(dev_stato, num_stato^1, rsp.get_number_of_species(), dev_context, 0);
	else
		Context<false><<< numBlocchi(rsp.get_number_of_species(), DIM_BLOCCO), DIM_BLOCCO>>>(dev_stato, num_stato^1, rsp.get_number_of_species(), dev_context, 0);

	// write on trace
	SaveTrace<<< numBlocchi(rsp.get_number_of_species(), DIM_BLOCCO), DIM_BLOCCO>>>( dev_stato, dev_trace, 0, rsp.get_number_of_species(), num_stato );

	*/

	for (unsigned int i=0; i<MAX_PASSI; i++) {
		/*
		printf(" * Switch state: %d.\n", num_stato);
		

		
				// dump a video dei risultati		
		hipMemcpy( read_back, dev_stato, sizeof(char)*rsp.get_number_of_species()*2, hipMemcpyDeviceToHost );
		hipDeviceSynchronize();
		for (unsigned int s=0; s<rsp.get_number_of_species()*2; s++) {
			printf("[Readback] Species %d state %d.\n", s, read_back[s+(num_stato^1)*rsp.get_number_of_species()]);
		}
		printf("\n");

		*/
		if (use_context) 
			Context<true><<< numBlocchi(rsp.get_number_of_species(), DIM_BLOCCO), DIM_BLOCCO>>>(dev_stato, num_stato^1, rsp.get_number_of_species(), dev_context, i);
		else
			Context<false><<< numBlocchi(rsp.get_number_of_species(), DIM_BLOCCO), DIM_BLOCCO>>>(dev_stato, num_stato^1, rsp.get_number_of_species(), dev_context, i);

		// Simulate<<< numBlocchi(rsp.get_number_of_reactions(), DIM_BLOCCO) , DIM_BLOCCO, 0, streams[selStream] >>>( dev_regole, dev_stato, dev_offset, num_stato, rsp.get_number_of_reactions(), rsp.get_number_of_species() );
		if (lightweight) {
			Simulate_Lightweight<true><<< numBlocchi(rsp.get_number_of_reactions(), DIM_BLOCCO) , DIM_BLOCCO >>>( dev_regole, dev_stato, dev_offset, num_stato, rsp.get_number_of_reactions(), rsp.get_number_of_species() );
		} else {
			if (use_constant)
				Simulate<true><<< numBlocchi(rsp.get_number_of_reactions(), DIM_BLOCCO) , DIM_BLOCCO >>>( dev_regole, dev_stato, dev_offset, num_stato, rsp.get_number_of_reactions(), rsp.get_number_of_species() );
			else
				Simulate<false><<< numBlocchi(rsp.get_number_of_reactions(), DIM_BLOCCO) , DIM_BLOCCO >>>( dev_regole, dev_stato, dev_offset, num_stato, rsp.get_number_of_reactions(), rsp.get_number_of_species() );
		}
		// write on trace
		SaveTrace<<< numBlocchi(rsp.get_number_of_species(), DIM_BLOCCO), DIM_BLOCCO>>>( dev_stato, dev_trace, i, rsp.get_number_of_species(), num_stato );

		hipDeviceSynchronize();

		// dump a video dei risultati		
		/*
		hipMemcpy( read_back, dev_stato, sizeof(char)*rsp.get_number_of_species()*2, hipMemcpyDeviceToHost );
		hipDeviceSynchronize();
		for (unsigned int s=0; s<rsp.get_number_of_species()*2; s++) {
			printf("[Readback] Species %d state %d.\n", s, read_back[s+(num_stato^1)*rsp.get_number_of_species()]);
		}
		printf("\n");
		*/
		
		
		num_stato ^= 1;			

		/*
		if (++selCount == statesPerGroup-1 ) {
			selCount = 0;

			unsigned long int copy_bytes = sizeof(char) * minimo( MAX_PASSI-(blockCount*statesPerGroup), statesPerGroup );			

			// memcpy asincrona
			hipMemcpyAsync( host_results[selCount], dev_results[selCount], copy_bytes, hipMemcpyDeviceToHost, streams[selStream]);
			
			selStream ^= 1;
			blockCount ++;
		}
		*/
		
	}

	stop_profiling(&start, &stop);

	hipMemcpy(host_trace, dev_trace, sizeof(char) * rsp.get_number_of_species() * MAX_PASSI, hipMemcpyDeviceToHost);

	if (!output_to_console) {
		std::ofstream output_file(output_path.c_str());
		if (output_file.is_open()) {

			output_file << "0" << "\t" ;

			for (unsigned int s=0; s<rsp.get_number_of_species(); s++) {
				if (rsp.vector_context[ s ]) {
					output_file << rsp.rev_insieme_specie[s]  << "\t";
				}
			}
			output_file << std::endl;

			for (unsigned int step=0; step<MAX_PASSI; step++) {
				output_file << step+1 << "\t";
				for (unsigned int species =0; species < rsp.get_number_of_species(); species++) {
					if ( host_trace[ rsp.get_number_of_species()*step + species ] == 1 ) {
						output_file << rsp.rev_insieme_specie[species] << "\t";
					}				
				}
				output_file << "\n";
			}
			output_file.close();
		} else {
			perror("ERROR: cannot save output dynamics to file.\n");
			exit(-1);
		}
	}  else { // output to console

		std::cout << "0 ";
		for (unsigned int s=0; s<rsp.get_number_of_species(); s++) {			
			if (rsp.vector_context[ s ]) {
				std::cout << rsp.rev_insieme_specie[s]  << " ";
			}
		}
		std::cout << std::endl;

		for (unsigned int step=0; step<MAX_PASSI; step++) {
			std::cout << step+1 << " ";
			for (unsigned int species =0; species < rsp.get_number_of_species(); species++) {
				if ( host_trace[ rsp.get_number_of_species()*step + species ] == 1 ) {
					std::cout << rsp.rev_insieme_specie[species] << " ";
				}				
			}
			std::cout << "\n";
		}

	}

	// system("pause");
    return 0;
}
